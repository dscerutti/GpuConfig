// -*-c++-*-
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <nvml.h>
#include <hiprand/hiprand_kernel.h>
#include <vector>

#include "Reporting/error_format.h"
#include "hpc_status.h"

namespace omni {
namespace cuda {

extern std::vector<GpuDetails> queryGpuStats() {

  // Test that there is a GPU in the system.  Initialize
  // a vector of specs to store all detected GPUs.
  int n_gpus;
  if (hipGetDeviceCount(&n_gpus) != hipSuccess) {
    if (n_gpus == 0) {
      rt_err("Error.  No CUDA-capable devices were found.", "queryGpuStats");
    }
  }
  std::vector<GpuDetails> device_catalog;
  device_catalog.resize(n_gpus);
  
  // Activate zero-copy
  if (hipSetDeviceFlags(hipDeviceMapHost) != hipSuccess) {
    rt_err("Unable to establish hipDeviceMapHost with hipSetDeviceFlags().", "queryGpuStats");
  }

  // Initialize the NVIDIA Management Library
  nvmlReturn_t nvml_init_result = nvmlInit_v2();
  if (nvml_init_result == NVML_ERROR_DRIVER_NOT_LOADED) {
    rt_err("nvmlInit_v2() failed with NVML_ERROR_DRIVER_NOT_LOADED.", "queryGpuStats");
  }
  else if (nvml_init_result == NVML_ERROR_NO_PERMISSION) {
    rt_err("nvmlInit_v2() failed with NVML_ERROR_NO_PERMISSION.", "queryGpuStats");
  }
  else if (nvml_init_result == NVML_ERROR_UNKNOWN) {
    rt_err("nvmlInit_v2() failed with NVML_ERROR_UNKNOWN.", "queryGpuStats");
  }

  // Get device properties
  for (int i = 0; i < n_gpus; i++) {
    std::vector<nvmlProcessInfo_t> nvml_info(32);
    nvmlDevice_t nt_device;
    hipDeviceProp_t device_properties;
    if (hipGetDeviceProperties(&device_properties, i) != hipSuccess) {
      printf("queryGpuStats :: Warning.  Unable to query properties for GPU %d\n", i);
      continue;
    }

    // Transcribe information about this GPU
    device_catalog[i].arch_major            = device_properties.major;
    device_catalog[i].arch_minor            = device_properties.minor;
    device_catalog[i].smp_count             = device_properties.multiProcessorCount;
    device_catalog[i].card_ram              = device_properties.totalGlobalMem;
    device_catalog[i].max_threads_per_block = device_properties.maxThreadsPerBlock;
    device_catalog[i].max_threads_per_smp   = device_properties.maxThreadsPerMultiProcessor;
    device_catalog[i].max_blocks_per_smp    = ;
    device_catalog[i].max_shared_per_block  = ;
    device_catalog[i].max_shared_per_smp    = ;
    device_catalog[i].registers_per_smp     = ;
    device_catalog[i].card_name             = std::string(device_properties.name);

    // Determine whether the GPU is available
    if (device_properties.major < 3) {
      device_catalog[i].available = false;
      device_catalog[i].supported = false;
    }
    else if (device_properties.major >= 3) {
      device_catalog[i].supported = true;
      if (nvmlDeviceGetHandleByIndex_v2(i, &nt_device) != NVML_SUCCESS) {
        printf("queryGpuStats :: Warning.  Unable to get device handle for GPU %d\n", i);
        continue;
      }
      unsigned int nvml_item_count = 0;
      nvmlReturn_t nv_status = nvmlDeviceGetComputeRunningProcesses(nt_device, &nvml_item_count,
								    nvml_info.data());
      if (nv_status != NVML_SUCCESS && nv_status != NVML_ERROR_INSUFFICIENT_SIZE) {
        printf("queryGpuStats :: Warning.  Unable to monitor activity on GPU %d [error %u]\n", i,
	       nv_status);
      }
      unsigned long long int mem_occ = 0;
      for (int j = 0; j < nvml_item_count; j++) {
        mem_occ += nvml_info[j].usedGpuMemory;
      }
      device_catalog[i].available = (mem_occ < significant_gpu_activity);
    }
  }

  // Shut down the NVIDIA Management Library
  if (nvmlShutdown() != NVML_SUCCESS) {
    rt_err("Error executing nvmlShutdown().", "queryGpuStats");
  }

  return device_catalog;
}

/// \brief Return the indices, specs, and one or more
void HpcStatus::getGpuDevice(int requested_count) {

  // Make a list of supported and available GPUs by their device indices
  std::vector<int> selections;
  for (int i = 0; i < gpu_count; i++) {
    if (gpu_list[i].available) {
      selections.push_back[i];
    }
  }
  
  // Select a device from the list
  if (hipSetValidDevices(selections.data(), ndev) != hipSuccess) {
    hipDeviceReset();
    rt_err("Error searching for compatible GPU.", "getGpuDevice");
  }

  // Establish the CUDA context
  if (hipFree(0) != hipSuccess) {
    hipDeviceReset();
    rt_err("Error initializing the CUDA context with hipFree(0).", "getGpuDevice");
  }

  // Get the device (this is a sanity check to ensure that the device can still be seen)
  int selected_device;
  if (hipGetDevice(&selected_device) != hipSuccess) {
    hipDeviceReset();
    rt_err("Error selecting GPU.", "getGpuDevice");
  }

  // Set the device so that it will be used in all future calculations launched by this thread
  if (hipSetDevice(&selected_device) != hipSuccess) {
    hipDeviceReset();
    rt_err("Error setting GPU.", "getGpuDevice");
  }  
}
  
} // namespace cuda
} // namespace omni
